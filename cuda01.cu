#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define N (1000*1024)

__global__ 
void vector_add(float *out, float *a, float *b, int n) {
    
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) out[i] = a[i] + b[i];

}

void printDevProp(hipDeviceProp_t devProp)
{
    printf("Major revision number:         %d\n",  devProp.major);
    printf("Minor revision number:         %d\n",  devProp.minor);
    printf("Name:                          %s\n",  devProp.name);
    printf("Total global memory:           %lu\n",  devProp.totalGlobalMem);
    printf("Total shared memory per block: %lu\n",  devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n",  devProp.regsPerBlock);
    printf("Warp size:                     %d\n",  devProp.warpSize);
    printf("Maximum memory pitch:          %lu\n",  devProp.memPitch);
    printf("Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);

    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);

    printf("Clock rate:                    %d\n",  devProp.clockRate);
    printf("Total constant memory:         %lu\n",  devProp.totalConstMem);
    printf("Texture alignment:             %lu\n",  devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
    printf("Cuda Cores:                    %d\n", devProp.multiProcessorCount * 64); // ver 7.5
    printf("Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
    return;
}

int main(){
    int ndev;

    float *a, *b;
    float *d_a, *d_b, *d_out; 

    // Allocate memory
    a = (float*)malloc(sizeof(float) * N);
    b = (float*)malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    hipGetDeviceCount(&ndev);
    printf("Devices: %d \n", ndev);

    hipDeviceProp_t pdev;
    for (int i = 0; i < ndev; i++) {
        hipGetDeviceProperties(&pdev, i);
        printDevProp(pdev);
    }

    // Alocate device memory
    hipMalloc((void**)&d_a, sizeof(float) * N);
    hipMalloc((void**)&d_b, sizeof(float) * N);
    hipMalloc((void**)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    
    // Main function
    vector_add<<<(N+256)/256, 256>>>(d_out, d_a, d_b, N);
}
